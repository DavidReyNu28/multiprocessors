#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void myKernel(void) {
	printf("GPU: Hello world!\n");
}


int main() {
	myKernel<<<2, 1>>>();
	hipDeviceSynchronize();
	printf("Ending...\n");
	return 0;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils/cheader.h"

#define SIZE 100000000

__global__ void add(int *c, int *a, int *b) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	while (i < SIZE) {
		c[i] = a[i] + b[i];
		i += blockDim.x * gridDim.x;
	}
}

int main(int argc, char* argv[]) {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	double ms;
	
	a = (int*) malloc(SIZE * sizeof(int));
	b = (int*) malloc(SIZE * sizeof(int));
	c = (int*) malloc(SIZE * sizeof(int));
	
	fill_array(a, SIZE);
	display_array("a:", a);
	fill_array(b, SIZE);
	display_array("b:", b);
	
	hipMalloc((void**) &d_a, SIZE * sizeof(int));
	hipMalloc((void**) &d_b, SIZE * sizeof(int));
	hipMalloc((void**) &d_c, SIZE * sizeof(int));
	
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	
	printf("Starting...\n");
	ms = 0;
	for (int i = 0; i < N; i++) {
		start_timer();
		add<<<128, 128>>>(d_c, d_a, d_b);
		ms += stop_timer();
	}
	
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	display_array("c:", c);
	printf("avg time = %.5lf ms\n", (ms / N));
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	free(a);
	free(b);
	free(c);
	
	return 0;
}
	

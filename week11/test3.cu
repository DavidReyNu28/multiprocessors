#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(int argc, char* argv[]) {
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc((void**) &d_a, size);
	hipMalloc((void**) &d_b, size);
	hipMalloc((void**) &d_c, size);

	scanf("%i %i", &a, &b);

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1, 1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	printf("c = %i\n", c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);	
	
	return 0;
}

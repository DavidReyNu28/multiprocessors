#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("Device name: %s\n", prop.name);
	}
	return 0;
}

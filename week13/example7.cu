#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils/cheader.h"

#define SIZE 8300000

__global__ void squares(int *a, int *b) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	b[i] = a[i] * a[i];
}

int main(int argc, char* argv[]) {
	int *a;
	int *d_a, *d_b;
	double ms;
	
	a = (int*) malloc(SIZE * sizeof(int));
	fill_array(a, SIZE);
	display_array("before:", a);
	
	hipMalloc((void**) &d_a, SIZE * sizeof(int));
	hipMalloc((void**) &d_b, SIZE * sizeof(int));
	
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	
	printf("Starting...\n");
	ms = 0;
	for (int i = 0; i < N; i++) {
		start_timer();
		squares<<<SIZE/128, 128>>>(d_a, d_b);
		ms += stop_timer();
	}
	
	hipMemcpy(a, d_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	display_array("after: ", a);
	printf("avg time O %.5lf ms\n", (ms/N) );
	
	hipFree(d_a);
	hipFree(d_b);
	
	free(a);
	
	return 0;
}

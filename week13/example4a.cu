#include "hip/hip_runtime.h"
/** Compile: nvcc example4.cu -lGL -lGLU -lglut **/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include "utils/cheader.h"
#include "utils/cpu_bitmap.h"

#define WIDTH	1024
#define HEIGHT	768
#define SCALEX	1.500
#define SCALEY	1.500

struct hipComplex {
	float real, img;
	
	__device__ hipComplex(float a, float b) : real(a), img(b) {}
	__device__ float magnitude2() {
		return (real * real) + (img * img);
	}
	__device__ hipComplex operator*(const hipComplex &a) {
		return hipComplex( ((real * a.real) - (img * a.img)) ,
						  ((img * a.real) + (real * a.img)) );
	}
	__device__ hipComplex operator+(const hipComplex &a) {
		return hipComplex( (real + a.real), (img + a.img) );
	}
};

enum color {RED, GREEN, BLUE, ALPHA};

__device__ int julia_value(int x, int y, int width, int height) {
	int k;
	float jx = SCALEX * (float) (width / 2 - x) / (width / 2);
	float jy = SCALEY * (float) (height / 2 - y) / (height / 2);
	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);
	
	for (k = 0; k < 200; k++) {
		a = a*a + c;
		if (a.magnitude2() > 1000) {
			return 0;
		}
	}
	return 1;
}

__global__ void julia_set(unsigned char *ptr, int width, int height) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	
	int value = julia_value(x, y, width, height);
	ptr[offset*4 + RED]   = (unsigned char) (255 * (0.4 * value));
	ptr[offset*4 + GREEN] = (unsigned char) (255 * (0.5 * value));
	ptr[offset*4 + BLUE]  = (unsigned char) (255 * (0.7 * value));
	ptr[offset*4 + ALPHA] = 255;
}

int main(int argc, char* argv[]) {
	CPUBitmap bitmap(WIDTH, HEIGHT);
	unsigned char *dev_bitmap;
	dim3 grid(WIDTH, HEIGHT);
	double ms;
	
	hipMalloc((void**) &dev_bitmap, bitmap.image_size());
	
	printf("Starting...\n");
	ms = 0;
	for (int i = 0; i < N; i++) {
		start_timer();
		julia_set<<<grid, 1>>>(dev_bitmap, WIDTH, HEIGHT);
		ms += stop_timer();
	}
	
	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
	hipFree(dev_bitmap);
	
	printf("avg time = %.5lf ms\n", (ms/N));
	bitmap.display_and_exit();
	return 0;
}
	
	
	
	
